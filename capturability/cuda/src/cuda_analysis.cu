#include "hip/hip_runtime.h"
#include "cuda_analysis.cuh"

namespace Cuda {

__host__ void outputBasin(std::string file_name, Condition cond, int *basin,
                          bool header) {
  FILE     *fp        = fopen(file_name.c_str(), "w");
  const int num_state = cond.grid->getNumState();

  // Header
  if (header) {
    fprintf(fp, "%s,", "state_id");
    fprintf(fp, "%s", "nstep");
    fprintf(fp, "\n");
  }

  // Data
  for (int state_id = 0; state_id < num_state; state_id++) {
    fprintf(fp, "%d,", state_id);
    fprintf(fp, "%d", basin[state_id]);
    fprintf(fp, "\n");
  }

  fclose(fp);
}

__host__ void outputNStep(std::string file_name, Condition cond, int *nstep, int *trans,
                          bool header) {
  FILE     *fp        = fopen(file_name.c_str(), "w");
  const int num_state = cond.grid->getNumState();
  const int num_input = cond.grid->getNumInput();
  int       max       = 0;

  // Header
  if (header) {
    fprintf(fp, "%s,", "state_id");
    fprintf(fp, "%s,", "input_id");
    fprintf(fp, "%s,", "trans");
    fprintf(fp, "%s", "nstep");
    fprintf(fp, "\n");
  }

  // Data
  int num_step[NUM_STEP_MAX + 1]; // 最大踏み出し歩数を10とする
  for(int i = 0; i < NUM_STEP_MAX + 1; i++) {
    num_step[i] = 0;
  }
  for (int state_id = 0; state_id < num_state; state_id++) {
    for (int input_id = 0; input_id < num_input; input_id++) {
      int id = state_id * num_input + input_id;
      fprintf(fp, "%d,", state_id);
      fprintf(fp, "%d,", input_id);
      fprintf(fp, "%d,", trans[id]);
      fprintf(fp, "%d", nstep[id]);
      fprintf(fp, "\n");
      if (max < nstep[id])
        max = nstep[id];
      if(nstep[id] > 0)
        num_step[nstep[id]]++;
    }
  }

  printf("max(nstep) = %d\n", max);
  for(int i = 1; i <= max; i++) {
    printf("%d-step capture point: %d\n", i, num_step[i]);
  }

  fclose(fp);
}

__host__ void outputCop(std::string file_name, Condition cond, Vector2 *cop,
                        bool header){
  FILE     *fp        = fopen(file_name.c_str(), "w");
  const int num_state = cond.grid->getNumState();

  // Header
  if (header) {
    fprintf(fp, "%s,", "state_id");
    fprintf(fp, "%s,", "cop_x");
    fprintf(fp, "%s", "cop_y");
    fprintf(fp, "\n");
  }

  // Data
  for(int state_id = 0; state_id < num_state; state_id++) {
    fprintf(fp, "%d,", state_id);
    fprintf(fp, "%lf,", cop[state_id].x_);
    fprintf(fp, "%lf", cop[state_id].y_);
    fprintf(fp, "\n");
  }

  fclose(fp);
}

__host__ void exeZeroStep(Capt::Grid grid, Capt::Model model, int *basin) {
  for (int state_id = 0; state_id < grid.getNumState(); state_id++) {
    Capt::State state = grid.getState(state_id);

    Capt::Polygon polygon;
    polygon.setVertex(model.getVec("foot", "foot_r_convex") );
    polygon.setVertex(model.getVec("foot", "foot_l_convex", state.swf) );

    bool flag = false;
    flag = polygon.inPolygon(state.icp, polygon.getConvexHull() );

    if (flag) {
      basin[state_id] = 0;
    }
  }
}

/* device function */

__device__ bool inPolygon(Vector2 point, Vector2 *vertex, int num_vertex){
  bool        flag    = false;
  double      product = 0.0;
  int         sign    = 0, on_line = 0;
  const float epsilon = 0.00001;

  for (size_t i = 0; i < num_vertex - 1; i++) {
    product = ( point - vertex[i] ) % ( vertex[i + 1] - vertex[i] );
    if (-epsilon <= product && product <= epsilon) {
      on_line += 1;
    } else if (product > 0) {
      sign += 1;
    } else if (product < 0) {
      sign -= 1;
    }
  }

  if (sign == int(num_vertex - 1 - on_line) ||
      sign == -int(num_vertex - 1 - on_line) ) {
    flag = true;
  }

  return flag;
}

__device__ Vector2 getClosestPoint(Vector2 point, Vector2* vertex, int num_vertex) {
  Vector2 closest;
  Vector2 v1, v2, v3, v4; // vector
  Vector2 n1, n2;         // normal vector

  if (inPolygon(point, vertex, num_vertex) ) {
    closest = point;
  } else {
    for (int i = 0; i < num_vertex - 1; i++) {
      //最近点が角にあるとき
      if (i == 0) {
        n1 = ( vertex[1] - vertex[i] ).normal();
        n2 = ( vertex[i] - vertex[num_vertex - 2] ).normal();
      } else {
        n1 = ( vertex[i + 1] - vertex[i] ).normal();
        n2 = ( vertex[i] - vertex[i - 1] ).normal();
      }
      v1 = point - vertex[i];
      if ( ( n1 % v1 ) < 0 && ( n2 % v1 ) > 0) {
        closest = vertex[i];
      }
      // 最近点が辺にあるとき
      n1 = ( vertex[i + 1] - vertex[i] ).normal();
      v1 = point - vertex[i];
      v2 = vertex[i + 1] - vertex[i];
      v3 = point - vertex[i + 1];
      v4 = vertex[i] - vertex[i + 1];
      if ( ( n1 % v1 ) > 0 && ( v2 % v1 ) < 0 && ( n1 % v3 ) < 0 && ( v4 % v3 ) > 0) {
        float k = v1 * v2 / ( v2.norm() * v2.norm() );
        closest = vertex[i] + k * v2;
      }
    }
  }

  return closest;
}

__device__ State step(State state, Input input, Vector2 cop, Physics *physics) {
  State state_;

  // 踏み出し時間
  Vector2 foot_dist = state.swf - input.swf;
  double  dist
    = sqrt(foot_dist.x() * foot_dist.x() + foot_dist.y() * foot_dist.y() );
  double t = dist / physics->v + physics->dt;

  // LIPM
  Vector2 icp = state.icp;
  icp = ( icp - cop ) * exp(physics->omega * t) + cop;

  // 状態変換
  state_.icp.setCartesian(-input.swf.x() + icp.x(), input.swf.y() - icp.y() );
  state_.swf.setCartesian(-input.swf.x(), input.swf.y() );

  return state_;
}

__device__ int roundValue(double value) {
  int integer = (int)value;

  double decimal = value - integer;
  if(decimal > 0) {
    if (decimal >= 0.5) {
      integer += 1;
    }
  }else{
    if (decimal <= -0.5) {
      integer -= 1;
    }
  }

  return integer;
}

__device__ bool existState(State state, GridCartesian *grid) {
  bool flag_icp_x = false, flag_icp_y = false;
  bool flag_swf_x = false, flag_swf_y = false;

  // icp_x
  if (state.icp.x_ >= grid->icp_x_min - grid->icp_x_step / 2.0 &&
      state.icp.x_ < grid->icp_x_max + grid->icp_x_step / 2.0) {
    flag_icp_x = true;
  }
  // icp_y
  if (state.icp.y_ >= grid->icp_y_min - grid->icp_y_step / 2.0 &&
      state.icp.y_ < grid->icp_y_max + grid->icp_y_step / 2.0) {
    flag_icp_y = true;
  }
  // swf_x
  if (state.swf.x_ >= grid->swf_x_min - grid->swf_x_step / 2.0 &&
      state.swf.x_ < grid->swf_x_max + grid->swf_x_step / 2.0) {
    flag_swf_x = true;
  }
  // swf_y
  if (state.swf.y_ >= grid->swf_y_min - grid->swf_y_step / 2.0 &&
      state.swf.y_ < grid->swf_y_max + grid->swf_y_step / 2.0) {
    flag_swf_y = true;
  }

  bool flag = flag_icp_x * flag_icp_y * flag_swf_x * flag_swf_y;
  return flag;
}

__device__ bool existState(State state, GridPolar *grid) {
  bool flag_icp_r = false, flag_icp_th = false;
  bool flag_swf_r = false, flag_swf_th = false;

  // icp_r
  if (state.icp.r_ >= grid->icp_r_min - grid->icp_r_step / 2.0 &&
      state.icp.r_ < grid->icp_r_max + grid->icp_r_step / 2.0) {
    flag_icp_r = true;
  }
  // icp_th
  if (state.icp.th_ >= grid->icp_th_min - grid->icp_th_step / 2.0 &&
      state.icp.th_ < grid->icp_th_max + grid->icp_th_step / 2.0) {
    flag_icp_th = true;
  }
  // swf_r
  if (state.swf.r_ >= grid->swf_r_min - grid->swf_r_step / 2.0 &&
      state.swf.r_ < grid->swf_r_max + grid->swf_r_step / 2.0) {
    flag_swf_r = true;
  }
  // swf_th
  if (state.swf.th_ >= grid->swf_th_min - grid->swf_th_step / 2.0 &&
      state.swf.th_ < grid->swf_th_max + grid->swf_th_step / 2.0) {
    flag_swf_th = true;
  }

  bool flag = flag_icp_r * flag_icp_th * flag_swf_r * flag_swf_th;
  return flag;
}

__device__ int getStateIndex(State state, GridCartesian *grid) {
  int icp_x_id = 0, icp_y_id = 0;
  int swf_x_id = 0, swf_y_id = 0;

  icp_x_id = roundValue( ( state.icp.x() - grid->icp_x_min ) / grid->icp_x_step);
  icp_y_id = roundValue( ( state.icp.y() - grid->icp_y_min ) / grid->icp_y_step);
  swf_x_id = roundValue( ( state.swf.x() - grid->swf_x_min ) / grid->swf_x_step);
  swf_y_id = roundValue( ( state.swf.y() - grid->swf_y_min ) / grid->swf_y_step);

  int state_id = 0;
  state_id = grid->swf_y_num * grid->swf_x_num * grid->icp_y_num * icp_x_id +
             grid->swf_y_num * grid->swf_x_num * icp_y_id +
             grid->swf_y_num * swf_x_id + swf_y_id;

  return state_id;
}

__device__ int getStateIndex(State state, GridPolar *grid) {
  int icp_r_id = 0, icp_th_id = 0;
  int swf_r_id = 0, swf_th_id = 0;

  icp_r_id  = roundValue( ( state.icp.r() - grid->icp_r_min ) / grid->icp_r_step);
  icp_th_id = roundValue( ( state.icp.th() - grid->icp_th_min ) / grid->icp_th_step);
  swf_r_id  = roundValue( ( state.swf.r() - grid->swf_r_min ) / grid->swf_r_step);
  swf_th_id = roundValue( ( state.swf.th() - grid->swf_th_min ) / grid->swf_th_step);

  int state_id = 0;
  state_id = grid->swf_th_num * grid->swf_r_num * grid->icp_th_num * icp_r_id +
             grid->swf_th_num * grid->swf_r_num * icp_th_id +
             grid->swf_th_num * swf_r_id + swf_th_id;

  return state_id;
}

/* global function */

__global__ void calcCop(State *state, GridCartesian *grid, Vector2 *foot, Vector2 *cop){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < grid->num_state) {
    cop[tid] = getClosestPoint(state[tid].icp, foot, grid->num_foot_vertex );

    tid += blockDim.x * gridDim.x;
  }
}

__global__ void calcTrans(State *state, Input *input, int *trans, GridCartesian *grid,
                          Vector2 *cop, Physics *physics){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < grid->num_state * grid->num_input) {
    int state_id = tid / grid->num_input;
    int input_id = tid % grid->num_input;

    State state_ = step(state[state_id], input[input_id], cop[state_id], physics);
    if (existState(state_, grid) )
      trans[tid] = getStateIndex(state_, grid);
    else
      trans[tid] = -1;

    tid += blockDim.x * gridDim.x;
  }
}

__global__ void calcTrans(State *state, Input *input, int *trans, GridPolar *grid,
                          Vector2 *cop, Physics *physics){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < grid->num_state * grid->num_input) {
    int state_id = tid / grid->num_input;
    int input_id = tid % grid->num_input;

    State state_ = step(state[state_id], input[input_id], cop[state_id], physics);
    if (existState(state_, grid) )
      trans[tid] = getStateIndex(state_, grid);
    else
      trans[tid] = -1;

    tid += blockDim.x * gridDim.x;
  }
}

__global__ void exeNStep(int N, int *basin,
                         int *nstep, int *trans, GridCartesian *grid) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < grid->num_state * grid->num_input) {
    int state_id = tid / grid->num_input;
    int input_id = tid % grid->num_input;

    if (trans[tid] >= 0) {
      if (basin[trans[tid]] == ( N - 1 ) ) {
        nstep[tid] = N;
        if (basin[state_id] < 0) {
          basin[state_id] = N;
        }
      }
    }

    tid += blockDim.x * gridDim.x;
  }
}

__global__ void exeNStep(int N, int *basin,
                         int *nstep, int *trans, GridPolar *grid) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < grid->num_state * grid->num_input) {
    int state_id = tid / grid->num_input;
    int input_id = tid % grid->num_input;

    if (trans[tid] >= 0) {
      if (basin[trans[tid]] == ( N - 1 ) ) {
        nstep[tid] = N;
        if (basin[state_id] < 0) {
          basin[state_id] = N;
        }
      }
    }

    tid += blockDim.x * gridDim.x;
  }
}

} // namespace Cuda