#include "capturability.h"

int main(void)
{
    hipSetDevice(1);
    //Xorgで使用しているGPUはkernel計算に制限時間があるため

    float cpR[N_CP_R], cpTh[N_CP_TH], stepR[N_FOOT_R], stepTh[N_FOOT_TH];
    linspace(cpR, CP_MIN_R, CP_MAX_R, N_CP_R);
    linspace(cpTh, CP_MIN_TH, CP_MAX_TH, N_CP_TH);
    linspace(stepR, FOOT_MIN_R, FOOT_MAX_R, N_FOOT_R);
    linspace(stepTh, FOOT_MIN_TH, FOOT_MAX_TH, N_FOOT_TH);

    float *dev_cpR, *dev_cpTh, *dev_stepR, *dev_stepTh;
    HANDLE_ERROR(hipMalloc((void **)&dev_cpR, N_CP_R*sizeof(float)));
    HANDLE_ERROR(hipMalloc((void **)&dev_cpTh, N_CP_TH*sizeof(float)));
    HANDLE_ERROR(hipMalloc((void **)&dev_stepR, N_FOOT_R*sizeof(float)));
    HANDLE_ERROR(hipMalloc((void **)&dev_stepTh, N_FOOT_TH*sizeof(float)));

    HANDLE_ERROR(hipMemcpy(dev_cpR, cpR, N_CP_R*sizeof(float),
                            hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_cpTh, cpTh, N_CP_TH*sizeof(float),
                            hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_stepR, stepR, N_FOOT_R*sizeof(float),
                            hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_stepTh, stepTh, N_FOOT_TH*sizeof(float),
                            hipMemcpyHostToDevice));

    Data *dataSet = new Data[N_STATE];
    initializing(dataSet, cpR, cpTh, stepR, stepTh);

    Data *dev_dataSet;
    HANDLE_ERROR(hipMalloc((void **)&dev_dataSet, N_STATE*sizeof(Data)));
    HANDLE_ERROR(hipMemcpy(dev_dataSet, dataSet, N_STATE*sizeof(Data),
                            hipMemcpyHostToDevice));

    step_1<<<BPG,TPB>>>(dev_dataSet);

    // 2-step Capturable Basin
    step_N<<<BPG,TPB>>>(dev_dataSet, 2, dev_cpR, dev_cpTh, dev_stepR, dev_stepTh);

    // 3-step Capturable Basin
    step_N<<<BPG,TPB>>>(dev_dataSet, 3, dev_cpR, dev_cpTh, dev_stepR, dev_stepTh);

    // 4-step Capturable Basin
    step_N<<<BPG,TPB>>>(dev_dataSet, 4, dev_cpR, dev_cpTh, dev_stepR, dev_stepTh);


    HANDLE_ERROR(hipMemcpy(dataSet, dev_dataSet, N_STATE*sizeof(Data),
    hipMemcpyDeviceToHost));

    writeData(dataSet, "data.csv");

    hipFree( dev_dataSet );
    hipFree( dev_cpR );
    hipFree( dev_cpTh );
    hipFree( dev_stepR );
    hipFree( dev_stepTh );

    delete [] dataSet;

    return 0;

}




















//
