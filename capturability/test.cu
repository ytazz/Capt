#include "hip/hip_runtime.h"
#include "cuda_analysis.cuh"
#include "grid.h"
#include "input.h"
#include "model.h"
#include "param.h"
#include "state.h"

const int BPG = 1024; // Blocks  Per Grid  (max: 65535)
const int TPB = 1024; // Threads Per Block (max: 1024)

using namespace CA;

int main(void) {
  /* パラメータの読み込み */
  Model model("nao.xml");
  Param param("analysis.xml");

  /* グリッド */
  Grid grid(param);
  const int num_state = grid.getNumState();
  const int num_input = grid.getNumInput();
  const int num_nstep = num_state * num_input;

  /* 足形状 */
  const int num_foot = (int)model.getVec("foot", "foot_r").size();
  // ホスト側
  CudaVector2 *cfoot_r = new CudaVector2[model.getVec("foot", "foot_r").size()];
  CudaVector2 *cfoot_l = new CudaVector2[model.getVec("foot", "foot_l").size()];
  for (size_t i = 0; i < model.getVec("foot", "foot_r").size(); i++) {
    cfoot_r[i].x_ = model.getVec("foot", "foot_r")[i].x;
    cfoot_r[i].y_ = model.getVec("foot", "foot_r")[i].y;
    cfoot_r[i].r_ = model.getVec("foot", "foot_r")[i].r;
    cfoot_r[i].th_ = model.getVec("foot", "foot_r")[i].th;
    cfoot_l[i].x_ = model.getVec("foot", "foot_l")[i].x;
    cfoot_l[i].y_ = model.getVec("foot", "foot_l")[i].y;
    cfoot_l[i].r_ = model.getVec("foot", "foot_l")[i].r;
    cfoot_l[i].th_ = model.getVec("foot", "foot_l")[i].th;
  }
  // デバイス側
  CudaVector2 *dev_cfoot_r;
  CudaVector2 *dev_cfoot_l;
  HANDLE_ERROR(
      hipMalloc((void **)&dev_cfoot_r, num_foot * sizeof(CudaVector2)));
  HANDLE_ERROR(
      hipMalloc((void **)&dev_cfoot_l, num_foot * sizeof(CudaVector2)));
  HANDLE_ERROR(hipMemcpy(dev_cfoot_r, cfoot_r, num_foot * sizeof(CudaVector2),
                          hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_cfoot_l, cfoot_l, num_foot * sizeof(CudaVector2),
                          hipMemcpyHostToDevice));

  /* グリッド */
  // ホスト側
  CudaState *cstate = new CudaState[num_state];
  CudaInput *cinput = new CudaInput[num_input];
  int *cnstep = new int[num_nstep];
  CudaGrid *cgrid = new CudaGrid;
  initNstep(grid, cnstep);
  copyState(grid, cstate);
  copyInput(grid, cinput);
  copyGrid(grid, model, param, cgrid);
  // デバイス側
  CudaState *dev_cstate;
  CudaInput *dev_cinput;
  int *dev_cnstep;
  CudaGrid *dev_cgrid;
  HANDLE_ERROR(hipMalloc((void **)&dev_cstate, num_state * sizeof(CudaState)));
  HANDLE_ERROR(hipMalloc((void **)&dev_cinput, num_input * sizeof(CudaInput)));
  HANDLE_ERROR(hipMalloc((void **)&dev_cnstep, num_nstep * sizeof(int)));
  HANDLE_ERROR(hipMalloc((void **)&dev_cgrid, sizeof(CudaGrid)));
  HANDLE_ERROR(hipMemcpy(dev_cstate, cstate, num_state * sizeof(CudaState),
                          hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_cinput, cinput, num_input * sizeof(CudaInput),
                          hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_cnstep, cnstep, num_nstep * sizeof(int),
                          hipMemcpyHostToDevice));
  HANDLE_ERROR(
      hipMemcpy(dev_cgrid, cgrid, sizeof(CudaGrid), hipMemcpyHostToDevice));

  exeZeroStep<<<BPG, TPB>>>(dev_cstate, dev_cinput, dev_cnstep, dev_cfoot_r,
                            dev_cfoot_l, dev_cgrid);

  HANDLE_ERROR(hipMemcpy(cnstep, dev_cnstep, num_nstep * sizeof(int),
                          hipMemcpyDeviceToHost));

  for (int i = 0; i < 4; i++) {
    printf("id: %d,\t nstep: %d\n", i, cnstep[i]);
  }

  /* メモリの開放 */
  // ホスト側
  delete cstate;
  delete cinput;
  delete cgrid;
  // デバイス側
  hipFree(dev_cstate);
  hipFree(dev_cinput);
  hipFree(dev_cnstep);
  hipFree(dev_cgrid);

  return 0;
}