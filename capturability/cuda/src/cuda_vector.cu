#include "hip/hip_runtime.h"
#include "cuda_vector.cuh"

__device__ CudaVector2::CudaVector2() { clear(); }

__device__ void CudaVector2::clear() {
  this->r_ = 0.0;
  this->th_ = 0.0;
  this->x_ = 0.0;
  this->y_ = 0.0;
}

__device__ void CudaVector2::setPolar(float radius, float theta) {
  this->r_ = radius;
  this->th_ = theta;
  polarToCartesian();
}

__device__ void CudaVector2::setCartesian(float x, float y) {
  this->x_ = x;
  this->y_ = y;
  cartesianToPolar();
}

__device__ void CudaVector2::polarToCartesian() {
  this->x_ = this->r_ * cos(this->th_);
  this->y_ = this->r_ * sin(this->th_);
}

__device__ void CudaVector2::cartesianToPolar() {
  this->r_ = norm();
  this->th_ = atan2f(this->y_, this->x_);
  if (this->th_ < 0.0) {
    this->th_ += 2 * M_PI;
  }
}

__device__ float CudaVector2::x() { return this->x_; }

__device__ float CudaVector2::y() { return this->y_; }

__device__ float CudaVector2::r() { return this->r_; }

__device__ float CudaVector2::th() { return this->th_; }

__device__ float CudaVector2::norm() { return sqrt(x_ * x_ + y_ * y_); }

__device__ CudaVector2 CudaVector2::normal() {
  CudaVector2 normal_vector;
  // rotate -90 deg around +z direction
  normal_vector.setCartesian(this->y_, -this->x_);
  return normal_vector;
}

__device__ CudaVector2 &CudaVector2::operator=(const CudaVector2 &v) {
  this->x_ = v.x_;
  this->y_ = v.y_;
  this->r_ = v.r_;
  this->th_ = v.th_;
  return *this;
}

__device__ CudaVector2 CudaVector2::operator+(const CudaVector2 &v) {
  CudaVector2 vec;
  double x, y;
  x = this->x_ + v.x_;
  y = this->y_ + v.y_;
  vec.setCartesian(x, y);
  return vec;
}

__device__ CudaVector2 CudaVector2::operator-(const CudaVector2 &v) {
  CudaVector2 vec;
  double x, y;
  x = this->x_ - v.x_;
  y = this->y_ - v.y_;
  vec.setCartesian(x, y);
  return vec;
}

__device__ float CudaVector2::operator%(const CudaVector2 &v) {
  float product;
  product = this->x_ * v.y_ - this->y_ * v.x_;
  return product;
}

__device__ CudaVector2 CudaVector2::operator*(const float &d) {
  CudaVector2 vec;
  double x, y;
  x = this->x_ * d;
  y = this->y_ * d;
  vec.setCartesian(x, y);
  return vec;
}

__device__ float CudaVector2::operator*(const CudaVector2 &v) {
  float product = 0.0;
  product += this->x_ * v.x_;
  product += this->y_ * v.y_;
  return product;
}

__device__ CudaVector2 CudaVector2::operator/(const float &d) {
  CudaVector2 vec;
  double x, y;
  x = this->x_ / d;
  y = this->y_ / d;
  vec.setCartesian(x, y);
  return vec;
}

__device__ CudaVector2 operator*(const float &d, const CudaVector2 &v) {
  CudaVector2 vec;
  double x, y;
  x = v.x_ * d;
  y = v.y_ * d;
  vec.setCartesian(x, y);
  return vec;
}