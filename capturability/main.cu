#include "cuda_analysis.cuh"
#include <chrono>

// Setting for Coordinate
typedef Cuda::GridCartesian grid_t;
// typedef Cuda::GridPolar grid_t;

int main(void) {
  std::chrono::system_clock::time_point start, end_exe, end_save;
  start = std::chrono::system_clock::now();

  /* 前処理 */
  printf("*** Analysis ***\n");
  printf("  Initializing ... ");
  fflush(stdout);

  /* パラメータの読み込み */
  Capt::Model cmodel("data/nao.xml");
  Capt::Param cparam("data/nao_xy.xml");
  Capt::Grid  cgrid(cparam);

  /* 各変数のsize */
  const int num_state  = cgrid.getNumState();
  const int num_input  = cgrid.getNumInput();
  const int num_grid   = num_state * num_input;
  const int num_foot_r = cmodel.getVec("foot", "foot_r_convex").size();
  const int num_foot_l = cmodel.getVec("foot", "foot_l_convex").size();
  const int num_vertex = num_foot_r + num_foot_l;
  const int num_swf    = cgrid.getNumInput();

  /* 解析条件 */
  Cuda::Condition cond;
  cond.model = &cmodel;
  cond.param = &cparam;
  cond.grid  = &cgrid;

  /* メモリ管理 */
  Cuda::MemoryManager mm;

  /* パラメータの用意 */
  // ホスト側
  Cuda::State   *state     = (Cuda::State*)malloc(sizeof( Cuda::State ) * num_state);
  Cuda::Input   *input     = (Cuda::Input*)malloc(sizeof( Cuda::Input ) * num_input );
  int           *trans     = (int*)malloc(sizeof( int ) * num_state * num_input );
  int           *basin     = (int*)malloc(sizeof( int ) * num_state );
  int           *nstep     = (int*)malloc(sizeof( int ) * num_grid );
  grid_t        *grid      = (grid_t*)malloc(sizeof( grid_t ) );
  Cuda::Vector2 *foot_r    = (Cuda::Vector2*)malloc(sizeof( Cuda::Vector2 ) * num_foot_r );
  Cuda::Vector2 *foot_l    = (Cuda::Vector2*)malloc(sizeof( Cuda::Vector2 ) * num_foot_l );
  Cuda::Vector2 *convex    = (Cuda::Vector2*)malloc(sizeof( Cuda::Vector2 ) * num_swf * num_vertex );
  Cuda::Vector2 *cop       = (Cuda::Vector2*)malloc(sizeof( Cuda::Vector2 ) * num_state );
  double        *step_time = (double*)malloc(sizeof( double ) * num_grid );
  Cuda::Physics *physics   = (Cuda::Physics*)malloc(sizeof( Cuda::Physics ) );
  mm.initHostState(state, cond);
  mm.initHostTrans(trans, cond);
  mm.initHostInput(input, cond);
  mm.initHostBasin(basin, cond);
  mm.initHostNstep(nstep, cond);
  mm.initHostGrid(grid, cond);
  mm.initHostFootR(foot_r, cond);
  mm.initHostFootL(foot_l, cond);
  mm.initHostConvex(convex, cond);
  mm.initHostCop(cop, cond);
  mm.initHostStepTime(step_time, cond);
  mm.initHostPhysics(physics, cond);
  mm.setGrid(grid);
  // デバイス側
  Cuda::State   *dev_state;
  Cuda::Input   *dev_input;
  int           *dev_basin;
  int           *dev_nstep;
  int           *dev_trans;
  grid_t        *dev_grid;
  Cuda::Vector2 *dev_foot_r;
  Cuda::Vector2 *dev_foot_l;
  Cuda::Vector2 *dev_convex;
  Cuda::Vector2 *dev_cop;
  double        *dev_step_time;
  Cuda::Physics *dev_physics;
  HANDLE_ERROR(hipMalloc( (void **)&dev_state, num_state * sizeof( Cuda::State ) ) );
  HANDLE_ERROR(hipMalloc( (void **)&dev_input, num_input * sizeof( Cuda::Input ) ) );
  HANDLE_ERROR(hipMalloc( (void **)&dev_trans, num_grid * sizeof( int ) ) );
  HANDLE_ERROR(hipMalloc( (void **)&dev_basin, num_state * sizeof( int ) ) );
  HANDLE_ERROR(hipMalloc( (void **)&dev_nstep, num_grid * sizeof( int ) ) );
  HANDLE_ERROR(hipMalloc( (void **)&dev_grid, sizeof( grid_t ) ) );
  HANDLE_ERROR(hipMalloc( (void **)&dev_foot_r, num_foot_r * sizeof( Cuda::Vector2 ) ) );
  HANDLE_ERROR(hipMalloc( (void **)&dev_foot_l, num_foot_l * sizeof( Cuda::Vector2 ) ) );
  HANDLE_ERROR(hipMalloc( (void **)&dev_convex, num_swf * num_vertex * sizeof( Cuda::Vector2 ) ) );
  HANDLE_ERROR(hipMalloc( (void **)&dev_cop, num_state * sizeof( Cuda::Vector2 ) ) );
  HANDLE_ERROR(hipMalloc( (void **)&dev_step_time, num_grid * sizeof( double ) ) );
  HANDLE_ERROR(hipMalloc( (void **)&dev_physics, sizeof( Cuda::Physics ) ) );
  // ホスト側からデバイス側にコピー
  mm.copyHostToDevState(state, dev_state);
  mm.copyHostToDevInput(input, dev_input);
  mm.copyHostToDevTrans(trans, dev_trans);
  mm.copyHostToDevBasin(basin, dev_basin);
  mm.copyHostToDevNstep(nstep, dev_nstep);
  mm.copyHostToDevGrid(grid, dev_grid);
  mm.copyHostToDevFootR(foot_r, dev_foot_r);
  mm.copyHostToDevFootL(foot_l, dev_foot_l);
  mm.copyHostToDevConvex(convex, dev_convex);
  mm.copyHostToDevCop(cop, dev_cop);
  mm.copyHostToDevStepTime(step_time, dev_step_time);
  mm.copyHostToDevPhysics(physics, dev_physics);

  printf("Done!\n");

  /* 状態遷移計算 */
  printf("  Calculating .... ");
  fflush(stdout);
  Cuda::calcCop << < BPG, TPB >> > ( dev_state, dev_grid, dev_foot_r, dev_cop );
  Cuda::calcStepTime << < BPG, TPB >> > ( dev_state, dev_input, dev_grid, dev_step_time, dev_physics );
  Cuda::calcBasin << < BPG, TPB >> > ( dev_state, dev_basin, dev_grid, dev_foot_r, dev_foot_l, dev_convex );
  Cuda::calcTrans << < BPG, TPB >> > ( dev_state, dev_input, dev_trans, dev_grid, dev_cop, dev_step_time, dev_physics );
  printf("Done!\n");

  /* 解析実行 */
  printf("  Analysing ...... ");
  fflush(stdout);

  int  step = 0;
  bool flag = true;
  while( flag ) {
    step++;
    Cuda::exeNstep << < BPG, TPB >> > ( step, dev_basin, dev_nstep, dev_trans, dev_grid );

    mm.copyDevToHostNstep(dev_nstep, nstep);
    flag = false;
    for(int id = 0; id < num_grid; id++) {
      if(nstep[id] == step)
        flag = true;
    }
  }
  end_exe = std::chrono::system_clock::now();
  printf("Done!\n");

  /* 解析結果をデバイス側からホスト側にコピー */
  mm.copyDevToHostBasin(dev_basin, basin);
  mm.copyDevToHostNstep(dev_nstep, nstep);
  mm.copyDevToHostTrans(dev_trans, trans);
  mm.copyDevToHostCop(dev_cop, cop);
  mm.copyDevToHostStepTime(dev_step_time, step_time);

  /* ファイル書き出し */
  Cuda::saveBasin("gpu/Basin.csv", cond, basin);
  Cuda::saveNStep("gpu/Nstep.csv", cond, nstep, trans);
  Cuda::saveCop("gpu/Cop.csv", cond, cop);
  Cuda::saveStepTime("gpu/StepTime.csv", cond, step_time);
  end_save = std::chrono::system_clock::now();

  /* 処理時間 */
  int time_exe  = std::chrono::duration_cast<std::chrono::milliseconds>(end_exe - start).count();
  int time_save = std::chrono::duration_cast<std::chrono::milliseconds>(end_save - end_exe).count();
  int time_sum  = std::chrono::duration_cast<std::chrono::milliseconds>(end_save - start).count();
  printf("*** Time ***\n");
  printf("  exe : %7d [ms]\n", time_exe);
  printf("  save: %7d [ms]\n", time_save);
  printf("  sum : %7d [ms]\n", time_sum);

  /* メモリの開放 */
  // ホスト側
  delete state;
  delete input;
  delete basin;
  delete nstep;
  delete trans;
  delete grid;
  delete foot_r;
  delete foot_l;
  delete convex;
  delete cop;
  delete step_time;
  delete physics;
  // デバイス側
  hipFree(dev_state);
  hipFree(dev_input);
  hipFree(dev_trans);
  hipFree(dev_basin);
  hipFree(dev_nstep);
  hipFree(dev_grid);
  hipFree(dev_foot_r);
  hipFree(dev_foot_l);
  hipFree(dev_convex);
  hipFree(dev_cop);
  hipFree(dev_step_time);
  hipFree(dev_physics);

  return 0;
}