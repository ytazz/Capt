#include "hip/hip_runtime.h"
#include "cuda_vector.cuh"

namespace Cuda {

__device__ void Vector2::clear() {
  this->x = 0.0;
  this->y = 0.0;
}

__device__ void Vector2::set(float x, float y) {
  this->x = x;
  this->y = y;
}

__device__ float Vector2::norm() {
  return sqrt(x * x + y * y);
}

__device__ Vector2 Vector2::normal() {
  Vector2 normal_vector;
  // rotate -90 deg around +z direction
  normal_vector.set(this->y, -this->x);
  return normal_vector;
}

__device__ Vector2 &Vector2::operator=(const Vector2 &v) {
  this->x = v.x;
  this->y = v.y;
  return *this;
}

__device__ Vector2 Vector2::operator+(const Vector2 &v) {
  Vector2 vec;
  vec.set(this->x + v.x, this->y + v.y);
  return vec;
}

__device__ Vector2 Vector2::operator-(const Vector2 &v) {
  Vector2 vec;
  vec.set(this->x - v.x, this->y - v.y);
  return vec;
}

__device__ float Vector2::operator%(const Vector2 &v) {
  float product;
  product = this->x * v.y - this->y * v.x;
  return product;
}

__device__ Vector2 Vector2::operator*(const float &d) {
  Vector2 vec;
  vec.set(this->x * d, this->y * d);
  return vec;
}

__device__ float Vector2::operator*(const Vector2 &v) {
  float product = this->x * v.x + this->y * v.y;
  return product;
}

__device__ Vector2 Vector2::operator/(const float &d) {
  Vector2 vec;
  vec.set(this->x / d, this->y / d);
  return vec;
}

__device__ Vector2 operator*(const float &d, const Vector2 &v) {
  Vector2 vec;
  vec.set(v.x * d, v.y * d);
  return vec;
}

__device__ void Vector3::clear() {
  this->x = 0.0;
  this->y = 0.0;
  this->z = 0.0;
}

__device__ void Vector3::set(float x, float y, float z) {
  this->x = x;
  this->y = y;
  this->z = z;
}

__device__ float Vector3::norm() {
  return sqrt(x * x + y * y + z * z);
}

__device__ Vector3 &Vector3::operator=(const Vector3 &v) {
  this->x = v.x;
  this->y = v.y;
  this->z = v.z;
  return *this;
}

__device__ Vector3 Vector3::operator+(const Vector3 &v) {
  Vector3 vec;
  vec.set(this->x + v.x, this->y + v.y, this->z + v.z);
  return vec;
}

__device__ Vector3 Vector3::operator-(const Vector3 &v) {
  Vector3 vec;
  vec.set(this->x - v.x, this->y - v.y, this->z - v.z);
  return vec;
}

__device__ Vector3 Vector3::operator*(const float &d) {
  Vector3 vec;
  vec.set(this->x * d, this->y * d, this->z * d);
  return vec;
}

__device__ float Vector3::operator*(const Vector3 &v) {
  float product = this->x * v.x + this->y * v.y + this->z * v.z;
  return product;
}

__device__ Vector3 Vector3::operator/(const float &d) {
  Vector3 vec;
  vec.set(this->x / d, this->y / d, this->z / d);
  return vec;
}

__device__ Vector3 operator*(const float &d, const Vector3 &v) {
  Vector3 vec;
  vec.set(v.x * d, v.y * d, v.z * d);
  return vec;
}

} // namespace Cuda