#include "capturability.h"

int main(void)
{
    float cpR[N_CP_R], cpTh[N_CP_TH], stepR[N_FOOT_R], stepTh[N_FOOT_TH];
    linspace(cpR, CP_MIN_R, CP_MAX_R, N_CP_R);
    linspace(cpTh, CP_MIN_TH, CP_MAX_TH, N_CP_TH);
    linspace(stepR, FOOT_MIN_R, FOOT_MAX_R, N_FOOT_R);
    linspace(stepTh, FOOT_MIN_TH, FOOT_MAX_TH, N_FOOT_TH);

    float *dev_cpR, *dev_cpTh, *dev_stepR, *dev_stepTh;
    HANDLE_ERROR(hipMalloc((void **)&dev_cpR, N_CP_R*sizeof(float)));
    HANDLE_ERROR(hipMalloc((void **)&dev_cpTh, N_CP_TH*sizeof(float)));
    HANDLE_ERROR(hipMalloc((void **)&dev_stepR, N_FOOT_R*sizeof(float)));
    HANDLE_ERROR(hipMalloc((void **)&dev_stepTh, N_FOOT_TH*sizeof(float)));
    HANDLE_ERROR(hipMemcpy(dev_cpR, cpR, N_CP_R*sizeof(float), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_cpTh, cpTh, N_CP_TH*sizeof(float), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_stepR, stepR, N_FOOT_R*sizeof(float), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_stepTh, stepTh, N_FOOT_TH*sizeof(float), hipMemcpyHostToDevice));

    States *statesSpace = new States[N_ENTIRE];
    makeStatesSpace(statesSpace, cpR, cpTh, stepR, stepTh);
    States *dev_StatesSpace;
    HANDLE_ERROR(hipMalloc((void **)&dev_StatesSpace, N_ENTIRE*sizeof(States)));
    HANDLE_ERROR(hipMemcpy(dev_StatesSpace, statesSpace, N_ENTIRE*sizeof(States),
                            hipMemcpyHostToDevice));

    PolarCoord *inputSpace = new PolarCoord[N_INPUT];
    makeInputSpace(inputSpace, stepR, stepTh);
    PolarCoord *dev_InputSpace;
    HANDLE_ERROR(hipMalloc((void **)&dev_InputSpace, N_INPUT*sizeof(PolarCoord)));
    HANDLE_ERROR(hipMemcpy(dev_InputSpace, inputSpace, N_INPUT*sizeof(PolarCoord),
                            hipMemcpyHostToDevice));

    // 0-step Capturable Basin
    step_0<<<BPG,TPB>>>(dev_StatesSpace);

    // 1-step Capturable Basin
    for (size_t i = 0; i < N_INPUT; i++) {
        step_N<<<BPG,TPB>>>(dev_StatesSpace, inputSpace[i], 1,
                            dev_cpR, dev_cpTh, dev_stepR, dev_stepTh);
    }

    // 2-step Capturable Basin
    for (size_t i = 0; i < N_INPUT; i++) {
        step_N<<<BPG,TPB>>>(dev_StatesSpace, inputSpace[i], 2,
                            dev_cpR, dev_cpTh, dev_stepR, dev_stepTh);
    }

    HANDLE_ERROR(hipMemcpy(statesSpace, dev_StatesSpace, N_ENTIRE*sizeof(States),
                            hipMemcpyDeviceToHost));


    std::vector<States> entire;
    for (size_t i = 0; i < N_ENTIRE; i++) {
        entire.push_back(statesSpace[i]);
    }

    writeFile(entire, "entire.csv");


    hipFree( dev_StatesSpace );
    hipFree( dev_InputSpace );
    hipFree( dev_cpR );
    hipFree( dev_cpTh );
    hipFree( dev_stepR );
    hipFree( dev_stepTh );

    delete [] statesSpace;
    delete [] inputSpace;


    return 0;

}
