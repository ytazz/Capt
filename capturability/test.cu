#include "hip/hip_runtime.h"
#include "cuda_analysis.cuh"
#include "grid.h"
#include "input.h"
#include "model.h"
#include "param.h"
#include "state.h"

const int BPG = 1024; // Blocks  Per Grid  (max: 65535)
const int TPB = 1024; // Threads Per Block (max: 1024)

using namespace CA;

int main(void) {
  /* 前処理 */
  /* ---------------------------------------------------------------------- */
  printf("Prepare...\t");

  /* パラメータの読み込み */
  Model model("nao.xml");
  Param param("analysis.xml");

  /* グリッド */
  Grid grid(param);
  const int num_state = grid.getNumState();
  const int num_input = grid.getNumInput();
  const int num_grid = num_state * num_input;

  /* 解析条件 */
  Condition cond;
  cond.model = &model;
  cond.param = &param;
  cond.grid = &grid;

  /* グリッド */
  // ホスト側
  CudaState *cstate = new CudaState[num_state];
  CudaInput *cinput = new CudaInput[num_input];
  int *cnstep = new int[num_grid];
  int *next_state_id = new int[num_grid];
  CudaGrid *cgrid = new CudaGrid;
  initState(cstate, next_state_id, cond);
  initInput(cinput, cond);
  initNstep(cnstep, cond);
  initGrid(cgrid, cond);
  // デバイス側
  CudaState *dev_cstate;
  CudaInput *dev_cinput;
  int *dev_cnstep;
  int *dev_next_state_id;
  CudaGrid *dev_cgrid;
  HANDLE_ERROR(hipMalloc((void **)&dev_cstate, num_state * sizeof(CudaState)));
  HANDLE_ERROR(hipMalloc((void **)&dev_cinput, num_input * sizeof(CudaInput)));
  HANDLE_ERROR(hipMalloc((void **)&dev_cnstep, num_grid * sizeof(int)));
  HANDLE_ERROR(hipMalloc((void **)&dev_next_state_id, num_grid * sizeof(int)));
  HANDLE_ERROR(hipMalloc((void **)&dev_cgrid, sizeof(CudaGrid)));
  HANDLE_ERROR(hipMemcpy(dev_cstate, cstate, num_state * sizeof(CudaState),
                          hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_cinput, cinput, num_input * sizeof(CudaInput),
                          hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_cnstep, cnstep, num_grid * sizeof(int),
                          hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_next_state_id, next_state_id,
                          num_grid * sizeof(int), hipMemcpyHostToDevice));
  HANDLE_ERROR(
      hipMemcpy(dev_cgrid, cgrid, sizeof(CudaGrid), hipMemcpyHostToDevice));

  /* CoP List */
  // ホスト側
  CudaVector2 *cop = new CudaVector2[num_state];
  initCop(cop, cond);
  // デバイス側
  CudaVector2 *dev_cop;
  HANDLE_ERROR(hipMalloc((void **)&dev_cop, num_state * sizeof(CudaVector2)));
  HANDLE_ERROR(hipMemcpy(dev_cop, cop, num_state * sizeof(CudaVector2),
                          hipMemcpyHostToDevice));

  printf("Done.\n");
  /* ------------------------------------------------------------------------ */

  /* 解析実行 */
  /* ---------------------------------------------------------------------- */
  printf("Execute...\t");

  exeZeroStep(grid, model, cnstep, next_state_id);
  // exeNStep<<<BPG, TPB>>>(dev_cstate, dev_cinput, dev_cnstep, dev_cfoot,
  //                           dev_cgrid);

  printf("Done.\n");
  /* ---------------------------------------------------------------------- */

  // HANDLE_ERROR(hipMemcpy(cnstep, dev_cnstep, num_grid * sizeof(int),
  //                         hipMemcpyDeviceToHost));

  /* ファイル書き出し */
  /* ---------------------------------------------------------------------- */
  printf("Output...\t");
  output("result.csv", cond, cnstep, next_state_id);
  printf("Done.\n");
  /* ---------------------------------------------------------------------- */

  /* 終了処理 */
  /* ---------------------------------------------------------------------- */
  printf("Finish...\t");

  /* メモリの開放 */
  // ホスト側
  delete cstate;
  delete cinput;
  delete cnstep;
  delete next_state_id;
  delete cgrid;
  // デバイス側
  hipFree(dev_cstate);
  hipFree(dev_cinput);
  hipFree(dev_next_state_id);
  hipFree(dev_cnstep);
  hipFree(dev_cgrid);

  printf("Done.\n");
  /* ---------------------------------------------------------------------- */

  return 0;
}