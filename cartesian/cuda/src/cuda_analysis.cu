#include "hip/hip_runtime.h"
#include "cuda_analysis.cuh"

namespace Cuda {

/* device function */

__device__ bool inPolygon(vec2_t point, vec2_t *convex, const int max_size, int swf_id){
  int num_vertex = 0;
  for(int i = 0; i < max_size; i++) {
    int convex_id = swf_id * max_size + i;
    if(convex[convex_id].x > -1) {
      num_vertex++;
    }
  }

  vec2_t *vertex = new vec2_t[num_vertex];
  for(int i = 0; i < num_vertex; i++) {
    int convex_id = swf_id * max_size + i;
    vertex[i] = convex[convex_id];
  }

  bool flag = inPolygon(point, vertex, num_vertex);

  delete vertex;

  return flag;
}

__device__ bool inPolygon(vec2_t point, vec2_t *vertex, int num_vertex){
  bool         flag    = false;
  double       product = 0.0;
  int          sign    = 0, on_line = 0;
  const double epsilon = 0.00001;

  for (size_t i = 0; i < num_vertex - 1; i++) {
    product = ( point - vertex[i] ) % ( vertex[i + 1] - vertex[i] );
    if (-epsilon <= product && product <= epsilon) {
      on_line += 1;
    } else if (product > 0) {
      sign += 1;
    } else if (product < 0) {
      sign -= 1;
    }
  }

  if (sign == int(num_vertex - 1 - on_line) ||
      sign == -int(num_vertex - 1 - on_line) ) {
    flag = true;
  }

  return flag;
}

__device__ vec2_t getClosestPoint(vec2_t point, vec2_t* vertex, int num_vertex) {
  vec2_t closest;
  vec2_t v1, v2, v3, v4; // vector
  vec2_t n1, n2;         // normal vector

  if (inPolygon(point, vertex, num_vertex) ) {
    closest = point;
  } else {
    for (int i = 0; i < num_vertex - 1; i++) {
      //最近点が角にあるとき
      if (i == 0) {
        n1 = ( vertex[1] - vertex[i] ).normal();
        n2 = ( vertex[i] - vertex[num_vertex - 2] ).normal();
      } else {
        n1 = ( vertex[i + 1] - vertex[i] ).normal();
        n2 = ( vertex[i] - vertex[i - 1] ).normal();
      }
      v1 = point - vertex[i];
      if ( ( n1 % v1 ) < 0 && ( n2 % v1 ) > 0) {
        closest = vertex[i];
      }
      // 最近点が辺にあるとき
      n1 = ( vertex[i + 1] - vertex[i] ).normal();
      v1 = point - vertex[i];
      v2 = vertex[i + 1] - vertex[i];
      v3 = point - vertex[i + 1];
      v4 = vertex[i] - vertex[i + 1];
      if ( ( n1 % v1 ) > 0 && ( v2 % v1 ) < 0 && ( n1 % v3 ) < 0 && ( v4 % v3 ) > 0) {
        double k = v1 * v2 / ( v2.norm() * v2.norm() );
        closest = vertex[i] + k * v2;
      }
    }
  }

  return closest;
}

__device__ State step(State state, Input input, double step_time, Physics *physics) {
  // LIPM
  vec2_t icp;
  icp = ( state.icp - input.cop ) * exp(physics->omega * step_time) + input.cop;

  // 状態変換
  State state_;
  state_.icp.set(-input.swf.x + icp.x, input.swf.y - icp.y );
  state_.swf.set(-input.swf.x, input.swf.y );
  state_.elp = 0;

  return state_;
}

__device__ int roundValue(double value) {
  int integer = (int)value;

  double decimal = value - integer;
  if(decimal > 0) {
    if (decimal >= 0.5) {
      integer += 1;
    }
  }else{
    if (decimal <= -0.5) {
      integer -= 1;
    }
  }

  return integer;
}

__device__ bool existState(State state, Grid *grid) {
  bool flag_icp_x = false, flag_icp_y = false;
  bool flag_swf_x = false, flag_swf_y = false;

  // icp_x
  if (state.icp.x >= grid->icp_x_min - grid->icp_x_stp / 2.0 &&
      state.icp.x < grid->icp_x_max + grid->icp_x_stp / 2.0) {
    flag_icp_x = true;
  }
  // icp_y
  if (state.icp.y >= grid->icp_y_min - grid->icp_y_stp / 2.0 &&
      state.icp.y < grid->icp_y_max + grid->icp_y_stp / 2.0) {
    flag_icp_y = true;
  }
  // swf_x
  if (state.swf.x >= grid->swf_x_min - grid->swf_x_stp / 2.0 &&
      state.swf.x < grid->swf_x_max + grid->swf_x_stp / 2.0) {
    flag_swf_x = true;
  }
  // swf_y
  if (state.swf.y >= grid->swf_y_min - grid->swf_y_stp / 2.0 &&
      state.swf.y < grid->swf_y_max + grid->swf_y_stp / 2.0) {
    flag_swf_y = true;
  }

  bool flag = flag_icp_x * flag_icp_y * flag_swf_x * flag_swf_y;
  return flag;
}

__device__ int getStateIndex(State state, Grid *grid) {
  int icp_x_id = 0, icp_y_id = 0;
  int swf_x_id = 0, swf_y_id = 0;
  int elp_t_id = 0;

  icp_x_id = roundValue( ( state.icp.x - grid->icp_x_min ) / grid->icp_x_stp);
  icp_y_id = roundValue( ( state.icp.y - grid->icp_y_min ) / grid->icp_y_stp);
  swf_x_id = roundValue( ( state.swf.x - grid->swf_x_min ) / grid->swf_x_stp);
  swf_y_id = roundValue( ( state.swf.y - grid->swf_y_min ) / grid->swf_y_stp);
  elp_t_id = 0;

  int state_id = 0;
  state_id = grid->elp_t_num * grid->swf_y_num * grid->swf_x_num * grid->icp_y_num * icp_x_id +
             grid->elp_t_num * grid->swf_y_num * grid->swf_x_num * icp_y_id +
             grid->elp_t_num * grid->swf_y_num * swf_x_id +
             grid->elp_t_num * swf_y_id +
             elp_t_id;

  return state_id;
}

/* global function */

__global__ void calcStepTime(State *state, Input *input, Grid *grid, double *step_time, Physics *physics){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  vec2_t foot_dist;
  while (tid < grid->state_num * grid->input_num) {
    int state_id = tid / grid->input_num;
    int input_id = tid % grid->input_num;

    // double tau  = max(0, step_time_min / 2 - state[state_id].elp) + dist / foot_vel_max + step_time_min / 2;
    foot_dist      = state[state_id].swf - input[input_id].swf;
    step_time[tid] = foot_dist.norm() / physics->v + physics->dt / 2;
    if(physics->dt / 2 - state[state_id].elp > 0) {
      step_time[tid] += physics->dt / 2 - state[state_id].elp;
    }

    tid += blockDim.x * gridDim.x;
  }
}

__global__ void calcBasin(State *state, int *basin, Grid *grid, vec2_t *foot_r, vec2_t *foot_l, vec2_t *convex){
  int       tid      = threadIdx.x + blockIdx.x * blockDim.x;
  const int max_size = grid->foot_r_num + grid->foot_l_num;

  if(enableDoubleSupport) {
    while (tid < grid->state_num) {
      int swf_id = tid % grid->input_num;
      // state[tid].elp < 0.001 means landing state, not swing phase
      if(inPolygon(state[tid].icp, convex, max_size, swf_id) && state[tid].elp < 0.001 )
        basin[tid] = 0;
      tid += blockDim.x * gridDim.x;
    }
  }else{
    while (tid < grid->state_num) {
      if(inPolygon(state[tid].icp, foot_r, grid->foot_r_num) && state[tid].elp < 0.001 )
        basin[tid] = 0;
      tid += blockDim.x * gridDim.x;
    }
  }
}

__global__ void calcTrans(State *state, Input *input, int *trans, Grid *grid,
                          double *step_time, Physics *physics){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < grid->state_num * grid->input_num) {
    int state_id = tid / grid->input_num;
    int input_id = tid % grid->input_num;

    State state_ = step(state[state_id], input[input_id], step_time[tid], physics);
    if (existState(state_, grid) )
      trans[tid] = getStateIndex(state_, grid);
    else
      trans[tid] = -1;

    tid += blockDim.x * gridDim.x;
  }
}

__global__ void exeNstep(int N, int *basin,
                         int *nstep, int *trans, Grid *grid) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < grid->state_num * grid->input_num) {
    int state_id = tid / grid->input_num;
    // int input_id = tid % grid->input_num;

    if (trans[tid] >= 0) {
      if (basin[trans[tid]] == ( N - 1 ) ) {
        nstep[tid] = N;
        if (basin[state_id] < 0) {
          basin[state_id] = N;
        }
      }
    }

    tid += blockDim.x * gridDim.x;
  }
}

} // namespace Cuda