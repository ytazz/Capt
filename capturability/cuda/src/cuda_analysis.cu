#include "hip/hip_runtime.h"
#include "cuda_analysis.cuh"

namespace Cuda {

__host__ void saveBasin(std::string file_name, Condition cond, int *basin,
                        bool header) {
  FILE     *fp        = fopen(file_name.c_str(), "w");
  const int num_state = cond.grid->getNumState();

  // Header
  if (header) {
    fprintf(fp, "%s,", "state_id");
    fprintf(fp, "%s", "nstep");
    fprintf(fp, "\n");
  }

  // Data
  for (int state_id = 0; state_id < num_state; state_id++) {
    fprintf(fp, "%d,", state_id);
    fprintf(fp, "%d", basin[state_id]);
    fprintf(fp, "\n");
  }

  fclose(fp);
}

__host__ void saveNStep(std::string file_name, Condition cond, int *nstep, int *trans,
                        bool header) {
  FILE     *fp        = fopen(file_name.c_str(), "w");
  const int num_state = cond.grid->getNumState();
  const int num_input = cond.grid->getNumInput();
  int       max       = 0;

  // Header
  if (header) {
    fprintf(fp, "%s,", "state_id");
    fprintf(fp, "%s,", "input_id");
    fprintf(fp, "%s,", "trans");
    fprintf(fp, "%s", "nstep");
    fprintf(fp, "\n");
  }

  // Data
  int num_step[NUM_STEP_MAX + 1]; // 最大踏み出し歩数を10とする
  for(int i = 0; i < NUM_STEP_MAX + 1; i++) {
    num_step[i] = 0;
  }
  for (int state_id = 0; state_id < num_state; state_id++) {
    for (int input_id = 0; input_id < num_input; input_id++) {
      int id = state_id * num_input + input_id;
      fprintf(fp, "%d,", state_id);
      fprintf(fp, "%d,", input_id);
      fprintf(fp, "%d,", trans[id]);
      fprintf(fp, "%d", nstep[id]);
      fprintf(fp, "\n");
      if (max < nstep[id])
        max = nstep[id];
      if(nstep[id] > 0)
        num_step[nstep[id]]++;
    }
  }

  printf("max(nstep) = %d\n", max);
  for(int i = 1; i <= max; i++) {
    printf("%d-step capture point: %d\n", i, num_step[i]);
  }

  fclose(fp);
}

__host__ void saveCop(std::string file_name, Condition cond, Vector2 *cop,
                      bool header){
  FILE     *fp        = fopen(file_name.c_str(), "w");
  const int num_state = cond.grid->getNumState();

  // Header
  if (header) {
    fprintf(fp, "%s,", "state_id");
    fprintf(fp, "%s,", "cop_x");
    fprintf(fp, "%s", "cop_y");
    fprintf(fp, "\n");
  }

  // Data
  for(int state_id = 0; state_id < num_state; state_id++) {
    fprintf(fp, "%d,", state_id);
    fprintf(fp, "%1.4lf,", cop[state_id].x_);
    fprintf(fp, "%1.4lf", cop[state_id].y_);
    fprintf(fp, "\n");
  }

  fclose(fp);
}

__host__ void saveStepTime(std::string file_name, Condition cond, double *step_time,
                           bool header){
  FILE     *fp        = fopen(file_name.c_str(), "w");
  const int num_state = cond.grid->getNumState();
  const int num_input = cond.grid->getNumInput();

  // Header
  if (header) {
    fprintf(fp, "%s,", "state_id");
    fprintf(fp, "%s,", "input_id");
    fprintf(fp, "%s", "step_time");
    fprintf(fp, "\n");
  }

  // Data
  for (int state_id = 0; state_id < num_state; state_id++) {
    for (int input_id = 0; input_id < num_input; input_id++) {
      int id = state_id * num_input + input_id;
      fprintf(fp, "%d,", state_id);
      fprintf(fp, "%d,", input_id);
      fprintf(fp, "%1.4lf", step_time[id]);
      fprintf(fp, "\n");
    }
  }

  fclose(fp);
}

/* device function */

__device__ bool inPolygon(Vector2 point, Vector2 *convex, const int max_size, int swf_id){
  int num_vertex = 0;
  for(int i = 0; i < max_size; i++) {
    int convex_id = swf_id * max_size + i;
    if(convex[convex_id].x_ > -1) {
      num_vertex++;
    }
  }

  Vector2 *vertex = new Vector2[num_vertex];
  for(int i = 0; i < num_vertex; i++) {
    int convex_id = swf_id * max_size + i;
    vertex[i] = convex[convex_id];
  }

  bool flag = inPolygon(point, vertex, num_vertex);

  delete vertex;

  return flag;
}

__device__ bool inPolygon(Vector2 point, Vector2 *vertex, int num_vertex){
  bool        flag    = false;
  double      product = 0.0;
  int         sign    = 0, on_line = 0;
  const float epsilon = 0.00001;

  for (size_t i = 0; i < num_vertex - 1; i++) {
    product = ( point - vertex[i] ) % ( vertex[i + 1] - vertex[i] );
    if (-epsilon <= product && product <= epsilon) {
      on_line += 1;
    } else if (product > 0) {
      sign += 1;
    } else if (product < 0) {
      sign -= 1;
    }
  }

  if (sign == int(num_vertex - 1 - on_line) ||
      sign == -int(num_vertex - 1 - on_line) ) {
    flag = true;
  }

  return flag;
}

__device__ Vector2 getClosestPoint(Vector2 point, Vector2* vertex, int num_vertex) {
  Vector2 closest;
  Vector2 v1, v2, v3, v4; // vector
  Vector2 n1, n2;         // normal vector

  if (inPolygon(point, vertex, num_vertex) ) {
    closest = point;
  } else {
    for (int i = 0; i < num_vertex - 1; i++) {
      //最近点が角にあるとき
      if (i == 0) {
        n1 = ( vertex[1] - vertex[i] ).normal();
        n2 = ( vertex[i] - vertex[num_vertex - 2] ).normal();
      } else {
        n1 = ( vertex[i + 1] - vertex[i] ).normal();
        n2 = ( vertex[i] - vertex[i - 1] ).normal();
      }
      v1 = point - vertex[i];
      if ( ( n1 % v1 ) < 0 && ( n2 % v1 ) > 0) {
        closest = vertex[i];
      }
      // 最近点が辺にあるとき
      n1 = ( vertex[i + 1] - vertex[i] ).normal();
      v1 = point - vertex[i];
      v2 = vertex[i + 1] - vertex[i];
      v3 = point - vertex[i + 1];
      v4 = vertex[i] - vertex[i + 1];
      if ( ( n1 % v1 ) > 0 && ( v2 % v1 ) < 0 && ( n1 % v3 ) < 0 && ( v4 % v3 ) > 0) {
        float k = v1 * v2 / ( v2.norm() * v2.norm() );
        closest = vertex[i] + k * v2;
      }
    }
  }

  return closest;
}

__device__ State step(State state, Input input, Vector2 cop, double step_time, Physics *physics) {
  // LIPM
  Vector2 icp;
  icp = ( state.icp - cop ) * exp(physics->omega * step_time) + cop;

  // 状態変換
  State state_;
  state_.icp.setCartesian(-input.swf.x() + icp.x(), input.swf.y() - icp.y() );
  state_.swf.setCartesian(-input.swf.x(), input.swf.y() );

  return state_;
}

__device__ int roundValue(double value) {
  int integer = (int)value;

  double decimal = value - integer;
  if(decimal > 0) {
    if (decimal >= 0.5) {
      integer += 1;
    }
  }else{
    if (decimal <= -0.5) {
      integer -= 1;
    }
  }

  return integer;
}

__device__ bool existState(State state, GridCartesian *grid) {
  bool flag_icp_x = false, flag_icp_y = false;
  bool flag_swf_x = false, flag_swf_y = false;

  // icp_x
  if (state.icp.x_ >= grid->icp_x_min - grid->icp_x_step / 2.0 &&
      state.icp.x_ < grid->icp_x_max + grid->icp_x_step / 2.0) {
    flag_icp_x = true;
  }
  // icp_y
  if (state.icp.y_ >= grid->icp_y_min - grid->icp_y_step / 2.0 &&
      state.icp.y_ < grid->icp_y_max + grid->icp_y_step / 2.0) {
    flag_icp_y = true;
  }
  // swf_x
  if (state.swf.x_ >= grid->swf_x_min - grid->swf_x_step / 2.0 &&
      state.swf.x_ < grid->swf_x_max + grid->swf_x_step / 2.0) {
    flag_swf_x = true;
  }
  // swf_y
  if (state.swf.y_ >= grid->swf_y_min - grid->swf_y_step / 2.0 &&
      state.swf.y_ < grid->swf_y_max + grid->swf_y_step / 2.0) {
    flag_swf_y = true;
  }

  bool flag = flag_icp_x * flag_icp_y * flag_swf_x * flag_swf_y;
  return flag;
}

__device__ bool existState(State state, GridPolar *grid) {
  bool flag_icp_r = false, flag_icp_th = false;
  bool flag_swf_r = false, flag_swf_th = false;

  // icp_r
  if (state.icp.r_ >= grid->icp_r_min - grid->icp_r_step / 2.0 &&
      state.icp.r_ < grid->icp_r_max + grid->icp_r_step / 2.0) {
    flag_icp_r = true;
  }
  // icp_th
  if (state.icp.th_ >= grid->icp_th_min - grid->icp_th_step / 2.0 &&
      state.icp.th_ < grid->icp_th_max + grid->icp_th_step / 2.0) {
    flag_icp_th = true;
  }
  // swf_r
  if (state.swf.r_ >= grid->swf_r_min - grid->swf_r_step / 2.0 &&
      state.swf.r_ < grid->swf_r_max + grid->swf_r_step / 2.0) {
    flag_swf_r = true;
  }
  // swf_th
  if (state.swf.th_ >= grid->swf_th_min - grid->swf_th_step / 2.0 &&
      state.swf.th_ < grid->swf_th_max + grid->swf_th_step / 2.0) {
    flag_swf_th = true;
  }

  bool flag = flag_icp_r * flag_icp_th * flag_swf_r * flag_swf_th;
  return flag;
}

__device__ int getStateIndex(State state, GridCartesian *grid) {
  int icp_x_id = 0, icp_y_id = 0;
  int swf_x_id = 0, swf_y_id = 0;

  icp_x_id = roundValue( ( state.icp.x() - grid->icp_x_min ) / grid->icp_x_step);
  icp_y_id = roundValue( ( state.icp.y() - grid->icp_y_min ) / grid->icp_y_step);
  swf_x_id = roundValue( ( state.swf.x() - grid->swf_x_min ) / grid->swf_x_step);
  swf_y_id = roundValue( ( state.swf.y() - grid->swf_y_min ) / grid->swf_y_step);

  int state_id = 0;
  state_id = grid->swf_y_num * grid->swf_x_num * grid->icp_y_num * icp_x_id +
             grid->swf_y_num * grid->swf_x_num * icp_y_id +
             grid->swf_y_num * swf_x_id + swf_y_id;

  return state_id;
}

__device__ int getStateIndex(State state, GridPolar *grid) {
  int icp_r_id = 0, icp_th_id = 0;
  int swf_r_id = 0, swf_th_id = 0;

  icp_r_id  = roundValue( ( state.icp.r() - grid->icp_r_min ) / grid->icp_r_step);
  icp_th_id = roundValue( ( state.icp.th() - grid->icp_th_min ) / grid->icp_th_step);
  swf_r_id  = roundValue( ( state.swf.r() - grid->swf_r_min ) / grid->swf_r_step);
  swf_th_id = roundValue( ( state.swf.th() - grid->swf_th_min ) / grid->swf_th_step);

  int state_id = 0;
  state_id = grid->swf_th_num * grid->swf_r_num * grid->icp_th_num * icp_r_id +
             grid->swf_th_num * grid->swf_r_num * icp_th_id +
             grid->swf_th_num * swf_r_id + swf_th_id;

  return state_id;
}

/* global function */

__global__ void calcCop(State *state, GridCartesian *grid, Vector2 *foot_r, Vector2 *cop){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < grid->num_state) {
    cop[tid] = getClosestPoint(state[tid].icp, foot_r, grid->num_foot_r );

    tid += blockDim.x * gridDim.x;
  }
}

__global__ void calcStepTime(State *state, Input *input, GridCartesian *grid, double *step_time, Physics *physics){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  Vector2 foot_dist;
  while (tid < grid->num_state * grid->num_input) {
    int state_id = tid / grid->num_input;
    int input_id = tid % grid->num_input;

    foot_dist      = state[state_id].swf - input[input_id].swf;
    step_time[tid] = foot_dist.norm() / physics->v + physics->dt;

    tid += blockDim.x * gridDim.x;
  }
}

__global__ void calcBasin(State *state, int *basin, GridCartesian *grid, Vector2 *foot_r, Vector2 *foot_l, Vector2 *convex){
  int       tid      = threadIdx.x + blockIdx.x * blockDim.x;
  const int max_size = grid->num_foot_r + grid->num_foot_l;

  if(enableDoubleSupport) {
    while (tid < grid->num_state) {
      int swf_id = tid % grid->num_input;

      if(inPolygon(state[tid].icp, convex, max_size, swf_id) )
        basin[tid] = 0;
      tid += blockDim.x * gridDim.x;
    }
  }else{
    while (tid < grid->num_state) {
      if(inPolygon(state[tid].icp, foot_r, grid->num_foot_r) )
        basin[tid] = 0;
      tid += blockDim.x * gridDim.x;
    }
  }
}

__global__ void calcTrans(State *state, Input *input, int *trans, GridCartesian *grid,
                          Vector2 *cop, double *step_time, Physics *physics){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < grid->num_state * grid->num_input) {
    int state_id = tid / grid->num_input;
    int input_id = tid % grid->num_input;

    State state_ = step(state[state_id], input[input_id], cop[state_id], step_time[tid], physics);
    if (existState(state_, grid) )
      trans[tid] = getStateIndex(state_, grid);
    else
      trans[tid] = -1;

    tid += blockDim.x * gridDim.x;
  }
}

__global__ void calcTrans(State *state, Input *input, int *trans, GridPolar *grid,
                          Vector2 *cop, double *step_time, Physics *physics){
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < grid->num_state * grid->num_input) {
    int state_id = tid / grid->num_input;
    int input_id = tid % grid->num_input;

    State state_ = step(state[state_id], input[input_id], cop[state_id], step_time[tid], physics);
    if (existState(state_, grid) )
      trans[tid] = getStateIndex(state_, grid);
    else
      trans[tid] = -1;

    tid += blockDim.x * gridDim.x;
  }
}

__global__ void exeNstep(int N, int *basin,
                         int *nstep, int *trans, GridCartesian *grid) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < grid->num_state * grid->num_input) {
    int state_id = tid / grid->num_input;
    int input_id = tid % grid->num_input;

    if (trans[tid] >= 0) {
      if (basin[trans[tid]] == ( N - 1 ) ) {
        nstep[tid] = N;
        if (basin[state_id] < 0) {
          basin[state_id] = N;
        }
      }
    }

    tid += blockDim.x * gridDim.x;
  }
}

__global__ void exeNstep(int N, int *basin,
                         int *nstep, int *trans, GridPolar *grid) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < grid->num_state * grid->num_input) {
    int state_id = tid / grid->num_input;
    int input_id = tid % grid->num_input;

    if (trans[tid] >= 0) {
      if (basin[trans[tid]] == ( N - 1 ) ) {
        nstep[tid] = N;
        if (basin[state_id] < 0) {
          basin[state_id] = N;
        }
      }
    }

    tid += blockDim.x * gridDim.x;
  }
}

} // namespace Cuda