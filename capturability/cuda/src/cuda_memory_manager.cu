#include "hip/hip_runtime.h"
#include "cuda_memory_manager.cuh"

namespace Cuda {

/* struct */

void GridCartesian::operator=(const GridCartesian &grid){
  this->num_state = grid.num_state;
  this->num_input = grid.num_input;
  this->num_grid  = grid.num_grid;

  this->icp_x_num = grid.icp_x_num;
  this->icp_y_num = grid.icp_y_num;
  this->swf_x_num = grid.swf_x_num;
  this->swf_y_num = grid.swf_y_num;

  this->icp_x_min  = grid.icp_x_min;
  this->icp_x_max  = grid.icp_x_max;
  this->icp_x_step = grid.icp_x_step;
  this->icp_y_min  = grid.icp_y_min;
  this->icp_y_max  = grid.icp_y_max;
  this->icp_y_step = grid.icp_y_step;
  this->swf_x_min  = grid.swf_x_min;
  this->swf_x_max  = grid.swf_x_max;
  this->swf_x_step = grid.swf_x_step;
  this->swf_y_min  = grid.swf_y_min;
  this->swf_y_max  = grid.swf_y_max;
  this->swf_y_step = grid.swf_y_step;
}

void GridPolar::operator=(const GridPolar &grid){
  this->num_state = grid.num_state;
  this->num_input = grid.num_input;
  this->num_grid  = grid.num_grid;

  this->icp_r_num  = grid.icp_r_num;
  this->icp_th_num = grid.icp_th_num;
  this->swf_r_num  = grid.swf_r_num;
  this->swf_th_num = grid.swf_th_num;

  this->icp_r_min   = grid.icp_r_min;
  this->icp_r_max   = grid.icp_r_max;
  this->icp_r_step  = grid.icp_r_step;
  this->icp_th_min  = grid.icp_th_min;
  this->icp_th_max  = grid.icp_th_max;
  this->icp_th_step = grid.icp_th_step;
  this->swf_r_min   = grid.swf_r_min;
  this->swf_r_max   = grid.swf_r_max;
  this->swf_r_step  = grid.swf_r_step;
  this->swf_th_min  = grid.swf_th_min;
  this->swf_th_max  = grid.swf_th_max;
  this->swf_th_step = grid.swf_th_step;
}

__device__ void State::operator=(const State &state) {
  this->icp = state.icp;
  this->swf = state.swf;
}

__device__ void Input::operator=(const Input &input) {
  this->swf = input.swf;
}

/* host function */

void MemoryManager::setGrid(GridCartesian* grid){
  this->grid.num_state = grid->num_state;
  this->grid.num_input = grid->num_input;
  this->grid.num_grid  = grid->num_grid;
}

void MemoryManager::setGrid(GridPolar* grid){
  this->grid.num_state = grid->num_state;
  this->grid.num_input = grid->num_input;
  this->grid.num_grid  = grid->num_grid;
}

__host__ void MemoryManager::initHostState(State *state, Condition cond) {
  const int num_state = cond.grid->getNumState();

  // state = (Cuda::State*)malloc(sizeof( Cuda::State ) * num_state);

  for (int state_id = 0; state_id < num_state; state_id++) {
    state[state_id].icp.x_  = cond.grid->getState(state_id).icp.x;
    state[state_id].icp.y_  = cond.grid->getState(state_id).icp.y;
    state[state_id].icp.r_  = cond.grid->getState(state_id).icp.r;
    state[state_id].icp.th_ = cond.grid->getState(state_id).icp.th;
    state[state_id].swf.x_  = cond.grid->getState(state_id).swf.x;
    state[state_id].swf.y_  = cond.grid->getState(state_id).swf.y;
    state[state_id].swf.r_  = cond.grid->getState(state_id).swf.r;
    state[state_id].swf.th_ = cond.grid->getState(state_id).swf.th;
  }
}

__host__ void MemoryManager::initHostInput(Input *input, Condition cond) {
  const int num_input = cond.grid->getNumInput();

  // input = (Cuda::Input*)malloc(sizeof( Cuda::Input ) * num_input );

  for (int input_id = 0; input_id < num_input; input_id++) {
    input[input_id].swf.x_  = cond.grid->getInput(input_id).swf.x;
    input[input_id].swf.y_  = cond.grid->getInput(input_id).swf.y;
    input[input_id].swf.r_  = cond.grid->getInput(input_id).swf.r;
    input[input_id].swf.th_ = cond.grid->getInput(input_id).swf.th;
  }
}

__host__ void MemoryManager::initHostTrans(int *trans, Condition cond) {
  const int num_state = cond.grid->getNumState();
  const int num_input = cond.grid->getNumInput();

  // trans = (int*)malloc(sizeof( int ) * num_state * num_input );

  for (int grid_id = 0; grid_id < num_state * num_input; grid_id++) {
    trans[grid_id] = -1;
  }
}

__host__ void MemoryManager::initHostBasin(int *basin, Condition cond) {
  const int num_state = cond.grid->getNumState();

  // basin = (int*)malloc(sizeof( int ) * num_state );

  for (int state_id = 0; state_id < num_state; state_id++) {
    basin[state_id] = -1;
  }
}

__host__ void MemoryManager::initHostNstep(int *nstep, Condition cond) {
  const int num_state = cond.grid->getNumState();
  const int num_input = cond.grid->getNumInput();
  const int num_grid  = num_state * num_input;

  // nstep = (int*)malloc(sizeof( int ) * num_grid );

  for (int grid_id = 0; grid_id < num_grid; grid_id++) {
    nstep[grid_id] = -1;
  }
}

__host__ void MemoryManager::initHostGrid(GridCartesian *grid, Condition cond) {
  // grid = new Cuda::GridCartesian;

  grid->num_state = cond.grid->getNumState();
  grid->num_input = cond.grid->getNumInput();
  grid->num_grid  = cond.grid->getNumState() * cond.grid->getNumInput();

  grid->icp_x_min  = cond.param->getVal("icp_x", "min");
  grid->icp_x_max  = cond.param->getVal("icp_x", "max");
  grid->icp_x_step = cond.param->getVal("icp_x", "step");
  grid->icp_x_num  = cond.param->getVal("icp_x", "num");

  grid->icp_y_min  = cond.param->getVal("icp_y", "min");
  grid->icp_y_max  = cond.param->getVal("icp_y", "max");
  grid->icp_y_step = cond.param->getVal("icp_y", "step");
  grid->icp_y_num  = cond.param->getVal("icp_y", "num");

  grid->swf_x_min  = cond.param->getVal("swf_x", "min");
  grid->swf_x_max  = cond.param->getVal("swf_x", "max");
  grid->swf_x_step = cond.param->getVal("swf_x", "step");
  grid->swf_x_num  = cond.param->getVal("swf_x", "num");

  grid->swf_y_min  = cond.param->getVal("swf_y", "min");
  grid->swf_y_max  = cond.param->getVal("swf_y", "max");
  grid->swf_y_step = cond.param->getVal("swf_y", "step");
  grid->swf_y_num  = cond.param->getVal("swf_y", "num");
}

__host__ void MemoryManager::initHostGrid(GridPolar *grid, Condition cond) {
  // grid = new Cuda::GridPolar;

  grid->num_state = cond.grid->getNumState();
  grid->num_input = cond.grid->getNumInput();
  grid->num_grid  = cond.grid->getNumState() * cond.grid->getNumInput();

  grid->icp_r_min  = cond.param->getVal("icp_r", "min");
  grid->icp_r_max  = cond.param->getVal("icp_r", "max");
  grid->icp_r_step = cond.param->getVal("icp_r", "step");
  grid->icp_r_num  = cond.param->getVal("icp_r", "num");

  grid->icp_th_min  = cond.param->getVal("icp_th", "min");
  grid->icp_th_max  = cond.param->getVal("icp_th", "max");
  grid->icp_th_step = cond.param->getVal("icp_th", "step");
  grid->icp_th_num  = cond.param->getVal("icp_th", "num");

  grid->swf_r_min  = cond.param->getVal("swf_r", "min");
  grid->swf_r_max  = cond.param->getVal("swf_r", "max");
  grid->swf_r_step = cond.param->getVal("swf_r", "step");
  grid->swf_r_num  = cond.param->getVal("swf_r", "num");

  grid->swf_th_min  = cond.param->getVal("swf_th", "min");
  grid->swf_th_max  = cond.param->getVal("swf_th", "max");
  grid->swf_th_step = cond.param->getVal("swf_th", "step");
  grid->swf_th_num  = cond.param->getVal("swf_th", "num");
}

__host__ void MemoryManager::initCop(Vector2 *cop, Condition cond){
  // cop = new Cuda::Vector2[grid.num_state];

  Capt::State                state;
  Capt::Polygon              polygon;
  std::vector<Capt::Vector2> region = cond.model->getVec("foot", "foot_r_convex");
  Capt::Vector2              cop_;

  for (int state_id = 0; state_id < cond.grid->getNumState(); state_id++) {
    state = cond.grid->getState(state_id);
    cop_  = polygon.getClosestPoint(state.icp, region);

    cop[state_id].x_  = cop_.x;
    cop[state_id].y_  = cop_.y;
    cop[state_id].r_  = cop_.r;
    cop[state_id].th_ = cop_.th;
  }
}

__host__ void MemoryManager::initPhysics(Physics *physics, Condition cond){
  // physics = new Cuda::Physics;

  physics->g     = cond.model->getVal("environment", "gravity");
  physics->h     = cond.model->getVal("physics", "com_height");
  physics->v     = cond.model->getVal("physics", "foot_vel_max");
  physics->dt    = cond.model->getVal("physics", "step_time_min");
  physics->omega = sqrt(physics->g / physics->h);
}

__host__ void MemoryManager::initDevState(Cuda::State *dev_state){
  HANDLE_ERROR(hipMalloc( (void **)&dev_state, grid.num_state * sizeof( Cuda::State ) ) );
}

__host__ void MemoryManager::initDevInput(Cuda::Input *dev_input){
  HANDLE_ERROR(hipMalloc( (void **)&dev_input, grid.num_input * sizeof( Cuda::Input ) ) );
}

__host__ void MemoryManager::initDevTrans(int *dev_trans){
  HANDLE_ERROR(hipMalloc( (void **)&dev_trans, grid.num_grid * sizeof( int ) ) );
}

__host__ void MemoryManager::initDevBasin(int *dev_basin){
  HANDLE_ERROR(hipMalloc( (void **)&dev_basin, grid.num_state * sizeof( int ) ) );
}

__host__ void MemoryManager::initDevNstep(int *dev_nstep){
  HANDLE_ERROR(hipMalloc( (void **)&dev_nstep, grid.num_grid * sizeof( int ) ) );
}

__host__ void MemoryManager::initDevGrid(Cuda::GridCartesian *dev_grid){
  HANDLE_ERROR(hipMalloc( (void **)&dev_grid, sizeof( Cuda::GridCartesian ) ) );
}

__host__ void MemoryManager::initDevGrid(Cuda::GridPolar *dev_grid){
  HANDLE_ERROR(hipMalloc( (void **)&dev_grid, sizeof( Cuda::GridPolar ) ) );
}

__host__ void MemoryManager::initDevCop(Vector2 *dev_cop){
  HANDLE_ERROR(hipMalloc( (void **)&dev_cop, grid.num_state * sizeof( Cuda::Vector2 ) ) );
}

__host__ void MemoryManager::initDevPhysics(Physics *dev_physics){
  HANDLE_ERROR(hipMalloc( (void **)&dev_physics, sizeof( Cuda::Physics ) ) );
}

__host__ void MemoryManager::copyHostToDevState(State *state, Cuda::State *dev_state){
  HANDLE_ERROR(hipMemcpy(dev_state, state, grid.num_state * sizeof( Cuda::State ), hipMemcpyHostToDevice) );
}

__host__ void MemoryManager::copyHostToDevInput(Input *input, Cuda::Input *dev_input){
  HANDLE_ERROR(hipMemcpy(dev_input, input, grid.num_input * sizeof( Cuda::Input ), hipMemcpyHostToDevice) );
}

__host__ void MemoryManager::copyHostToDevTrans(int *trans, int *dev_trans){
  HANDLE_ERROR(hipMemcpy(dev_trans, trans, grid.num_grid * sizeof( int ), hipMemcpyHostToDevice) );
}

__host__ void MemoryManager::copyHostToDevBasin(int *basin, int *dev_basin){
  HANDLE_ERROR(hipMemcpy(dev_basin, basin, grid.num_state * sizeof( int ), hipMemcpyHostToDevice) );
}

__host__ void MemoryManager::copyHostToDevNstep(int *nstep, int *dev_nstep){
  HANDLE_ERROR(hipMemcpy(dev_nstep, nstep, grid.num_grid * sizeof( int ), hipMemcpyHostToDevice) );
}

__host__ void MemoryManager::copyHostToDevGrid(GridCartesian *grid, Cuda::GridCartesian *dev_grid){
  HANDLE_ERROR(hipMemcpy(dev_grid, grid, sizeof( Cuda::GridCartesian ), hipMemcpyHostToDevice) );
}

__host__ void MemoryManager::copyHostToDevGrid(GridPolar *grid, Cuda::GridPolar *dev_grid){
  HANDLE_ERROR(hipMemcpy(dev_grid, grid, sizeof( Cuda::GridPolar ), hipMemcpyHostToDevice) );
}

__host__ void MemoryManager::copyHostToDevCop(Vector2 *cop, Vector2 *dev_cop){
  HANDLE_ERROR(hipMemcpy(dev_cop, cop, grid.num_state * sizeof( Cuda::Vector2 ), hipMemcpyHostToDevice ) );
}

__host__ void MemoryManager::copyHostToDevPhysics(Physics *physics, Physics *dev_physics){
  HANDLE_ERROR(hipMemcpy(dev_physics, physics, sizeof( Cuda::Physics ), hipMemcpyHostToDevice ) );
}

__host__ void MemoryManager::copyDevToHostState(Cuda::State *dev_state, State *state){
  HANDLE_ERROR(hipMemcpy(state, dev_state, grid.num_state * sizeof( Cuda::State ), hipMemcpyDeviceToHost) );
}

__host__ void MemoryManager::copyDevToHostInput(Cuda::Input *dev_input, Input *input){
  HANDLE_ERROR(hipMemcpy(input, dev_input, grid.num_input * sizeof( Cuda::Input ), hipMemcpyDeviceToHost) );
}

__host__ void MemoryManager::copyDevToHostTrans(int *dev_trans, int *trans){
  HANDLE_ERROR(hipMemcpy(trans, dev_trans, grid.num_grid * sizeof( int ), hipMemcpyDeviceToHost) );
}

__host__ void MemoryManager::copyDevToHostBasin(int *dev_basin, int *basin){
  HANDLE_ERROR(hipMemcpy(basin, dev_basin, grid.num_state * sizeof( int ), hipMemcpyDeviceToHost) );
}

__host__ void MemoryManager::copyDevToHostNstep(int *dev_nstep, int *nstep){
  HANDLE_ERROR(hipMemcpy(nstep, dev_nstep, grid.num_grid * sizeof( int ), hipMemcpyDeviceToHost) );
}

__host__ void MemoryManager::copyDevToHostGrid(Cuda::GridCartesian *dev_grid, GridCartesian *grid){
  HANDLE_ERROR(hipMemcpy(grid, dev_grid, sizeof( Cuda::GridCartesian ), hipMemcpyDeviceToHost) );
}

__host__ void MemoryManager::copyDevToHostGrid(Cuda::GridPolar *dev_grid, GridPolar *grid){
  HANDLE_ERROR(hipMemcpy(grid, dev_grid, sizeof( Cuda::GridPolar ), hipMemcpyDeviceToHost) );
}

} // namespace Cuda