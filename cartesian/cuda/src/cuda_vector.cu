#include "hip/hip_runtime.h"
#include "cuda_vector.cuh"

namespace Cuda {

__device__ void Vector2::clear() {
  this->x = 0.0;
  this->y = 0.0;
}

__device__ void Vector2::set(double x, double y) {
  this->x = x;
  this->y = y;
}

__device__ double Vector2::norm() {
  return sqrt(x * x + y * y);
}

__device__ Vector2 Vector2::normal() {
  Vector2 normal_vector;
  // rotate -90 deg around +z direction
  normal_vector.set(this->y, -this->x);
  return normal_vector;
}

__device__ Vector2 &Vector2::operator=(const Vector2 &v) {
  this->x = v.x;
  this->y = v.y;
  return *this;
}

__device__ Vector2 Vector2::operator+(const Vector2 &v) {
  Vector2 vec;
  vec.set(this->x + v.x, this->y + v.y);
  return vec;
}

__device__ Vector2 Vector2::operator-(const Vector2 &v) {
  Vector2 vec;
  vec.set(this->x - v.x, this->y - v.y);
  return vec;
}

__device__ double Vector2::operator%(const Vector2 &v) {
  double product;
  product = this->x * v.y - this->y * v.x;
  return product;
}

__device__ Vector2 Vector2::operator*(const double &d) {
  Vector2 vec;
  vec.set(this->x * d, this->y * d);
  return vec;
}

__device__ double Vector2::operator*(const Vector2 &v) {
  double product = this->x * v.x + this->y * v.y;
  return product;
}

__device__ Vector2 Vector2::operator/(const double &d) {
  Vector2 vec;
  vec.set(this->x / d, this->y / d);
  return vec;
}

__device__ Vector2 operator*(const double &d, const Vector2 &v) {
  Vector2 vec;
  vec.set(v.x * d, v.y * d);
  return vec;
}

__device__ void Vector3::clear() {
  this->x = 0.0;
  this->y = 0.0;
  this->z = 0.0;
}

__device__ void Vector3::set(double x, double y, double z) {
  this->x = x;
  this->y = y;
  this->z = z;
}

__device__ double Vector3::norm() {
  return sqrt(x * x + y * y + z * z);
}

__device__ Vector3 &Vector3::operator=(const Vector3 &v) {
  this->x = v.x;
  this->y = v.y;
  this->z = v.z;
  return *this;
}

__device__ Vector3 Vector3::operator+(const Vector3 &v) {
  Vector3 vec;
  vec.set(this->x + v.x, this->y + v.y, this->z + v.z);
  return vec;
}

__device__ Vector3 Vector3::operator-(const Vector3 &v) {
  Vector3 vec;
  vec.set(this->x - v.x, this->y - v.y, this->z - v.z);
  return vec;
}

__device__ Vector3 Vector3::operator*(const double &d) {
  Vector3 vec;
  vec.set(this->x * d, this->y * d, this->z * d);
  return vec;
}

__device__ double Vector3::operator*(const Vector3 &v) {
  double product = this->x * v.x + this->y * v.y + this->z * v.z;
  return product;
}

__device__ Vector3 Vector3::operator/(const double &d) {
  Vector3 vec;
  vec.set(this->x / d, this->y / d, this->z / d);
  return vec;
}

__device__ Vector3 operator*(const double &d, const Vector3 &v) {
  Vector3 vec;
  vec.set(v.x * d, v.y * d, v.z * d);
  return vec;
}

} // namespace Cuda